// incrementArray.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

void incrementArrayOnHost(float *a, int N)
{
  int i;
  for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int id  = blockIdx.y*gridDim.x*blockDim.x + idx;
  //printf("block: (%d,%d), thread:(%d), id: %d, val: %.6f\n", blockIdx.x, blockIdx.y, idx, id, a[id]);
  if (id<N) a[id] = a[id]+1.f;
}

int main(int argc, char **argv)
{
  float *a_h, *b_h; // pointers to host memory
  float *a_d; // pointer to device memory
  int i, N = atoi(argv[1]);
  size_t size = N*sizeof(float);

  // allocate arrays on host
  a_h = (float *)malloc(size);
  b_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &a_d, size);
  // initialization of host data
  for (i=0; i<N; i++) a_h[i] = (float)i;
  // copy data from host to device
  hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
  
  // do calculation on host
  incrementArrayOnHost(a_h, N);

  // do calculation on device:
  // Part 1 of 2. Compute execution configuration
  int gridDimX = atoi(argv[2]);
  int gridDimY = atoi(argv[3]);
  int gridDim = gridDimX * gridDimY;
  dim3 blockSize = dim3(N/gridDim + ((N%gridDim)?1:0));
  dim3 gridSize = dim3(gridDimX,gridDimY);
  printf("N: %d, nBlocks: %d\n", N, gridDim);

  // Part 2 of 2. Call incrementArrayOnDevice kernel
  incrementArrayOnDevice <<< gridSize, blockSize >>> (a_d, N);
  // Retrieve result from device and store in b_h
  hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  // check results
  for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);

  // cleanup
  free(a_h); free(b_h); hipFree(a_d);
}
