// incrementArray.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

void incrementArrayOnHost(float *a, int N)
{
  int i;
  for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id  = idy*gridDim.x*blockDim.x + idx;
  //printf("block: (%d,%d), thread:(%d), id: %d, val: %.6f\n", blockIdx.x, blockIdx.y, idx, id, a[id]);
  if (id<N) a[id] = a[id]+1.f;
}

int main(int argc, char **argv)
{
  float *a_h, *b_h; // pointers to host memory
  float *a_d; // pointer to device memory
  int i, N = atoi(argv[1]);
  size_t size = N*sizeof(float);

  // allocate arrays on host
  a_h = (float *)malloc(size);
  b_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &a_d, size);
  // initialization of host data
  for (i=0; i<N; i++) a_h[i] = (float)i;
  // copy data from host to device
  hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
  
  // do calculation on host
  incrementArrayOnHost(a_h, N);

  // do calculation on device:
  // Part 1 of 2. Compute execution configuration
  int blockDimX = atoi(argv[2]);
  int blockDimY = atoi(argv[3]);
  int gridDimX = atoi(argv[4]);
  int gridDimY = atoi(argv[5]);

  dim3 blockSize = dim3(blockDimX, blockDimY);
  dim3 gridSize = dim3(gridDimX, gridDimY);
  printf("blockDim: (%d,%d), gridDim: (%d,%d)\n", blockDimX, blockDimY, gridDimX, gridDimY);

  // Part 2 of 2. Call incrementArrayOnDevice kernel
  incrementArrayOnDevice <<< gridSize, blockSize >>> (a_d, N);
  // Retrieve result from device and store in b_h
  hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  // check results
  for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);

  // cleanup
  free(a_h); free(b_h); hipFree(a_d);
}
