#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "helper.c"

#define MAX_LEN 1000

__global__ void calculateDistances(hipblasHandle_t handle, int m, int n, float **X_train, float *row_test, float *distances)
{
  int id = blockIdx.x*gridDim.x + threadIdx.x;
  float al = -1.0;
  float dist;
  hipblasSaxpy(handle, n, &al, X_train[id], 1, row_test, 1);
  hipblasSnrm2(handle, n, row_test, 1, &dist);
  distances[id] = dist;
}

void getTopKRows(int m, int k, float *distances, int *indices)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int idxMin, idxMax;
  float maxVal;
  hipblasIsamax(handle, m, distances, 1, &idxMax);
  hipDeviceSynchronize();
  maxVal = distances[idxMax-1];
  for (int i=0; i<k; i++)
  {
    hipblasIsamin(handle, m, distances, 1, &idxMin);
    hipDeviceSynchronize();
    indices[i] = idxMin-1;
    distances[idxMin-1] = maxVal;
  }

  hipblasDestroy(handle);
}

int main(int argc, char **argv)
{
  int m = 150;
  int t = 3;
  int n = 5;
  
  float **X_train = mallocUniX(m, n-1), **X_test = mallocUniX(t, n-1);
  float *y_train = mallocUniY(m), *y_test = mallocUniY(t), *distances = mallocUniY(m);

  char **data_train = mallocData(m, n, MAX_LEN);
  char **data_test = mallocData(t, n, MAX_LEN);
  read_csv(m, n, data_train, "test_input/Iris_150.csv");
  read_csv(t, n, data_test, "test_input/Iris_test.csv");
  getXandY(m, n, data_train, X_train, y_train);
  getXandY(t, n, data_test, X_test, y_test);
  freeData(m, n, data_train);
  freeData(t, n, data_test);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  calculateDistances<<<1,150>>>(handle, m, n, X_train, X_test[0], distances);
  hipDeviceSynchronize();
  hipblasDestroy(handle);  

  freeUniX(m, X_train); freeUniX(t, X_test);
  freeUniY(y_train); freeUniY(y_test);

  /*
  int m = atoi(argv[1]);
  int n = atoi(argv[2]);
  int k = atoi(argv[3]);

  float *X_train, *X_test, *distances;
  hipMallocManaged((void **)&X_train, m*n*sizeof(float));
  hipMallocManaged((void **)&X_test, m*n*sizeof(float));
  hipMallocManaged((void **)&X_test, m*sizeof(float));
  for (int i=0; i<m; i++)
  {
    for (int j=0; j<n; j++)
    {
      X_train[i*n+j] = i*n+j;
      X_test[i*n+j] = i*n+j+1;
    }
  }

  hipFree(X_train); hipFree(X_test);
  */
  return 0;
}
