#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hipblas.h"
#include "tools.c"

#define MAX_LEN 1000
#define BLOCK_SIZE 512

__global__ void getSquaredNorm(int m, int n, float *X, float *normX)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < m)
  {
    float norm = 0.0;
    for (int j=0; j<n; j++)
    {
      float val = X[idx*n+j];
      norm += (val*val);
    }
    normX[idx] = norm;
  }
}

__global__ void calculateDistance(int m, int t, float *normX, float *normY, float *xTy)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < m*t)
  {
    int i = idx/m, j = idx%m;
    float squaredDist = normX[j] + normY[i] + xTy[idx];
    float dist = sqrt(squaredDist);
    xTy[idx] = dist;
  }
}

__global__ void parallel_sort(int t, int m, int k, float *matrix, int *result)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < t)
  {
    for (int i=0; i<k; i++)
    {
      int min_idx = -1, min_idx_g = -1;
      float min_val;
      for (int j=0; j<m; j++)
      {
        int valid = 1;
        for (int jj=0; jj<i; jj++) valid = valid && (j != result[idx*k+jj]);
        if (valid)
        {
          float val = matrix[idx*m+j];
          if ((min_idx == -1) || (val < min_val))
          {
            min_val = val;
            min_idx = idx*m+j;
            min_idx_g = j;
          }
        }
      }
      result[idx*k+i] = min_idx_g;
    }
  }
}

void matmulTrainTest(int m, int n, int k, float *A, float *B, float *C)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float al = -2.0f, bet = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &al, B, k, A, k, &bet, C, n);
  hipblasDestroy(handle); 
}

int main(int argc, char **argv)
{
  int m = atoi(argv[1]);
  int t = atoi(argv[2]);
  int n = atoi(argv[3]);
  int k = atoi(argv[4]);
  int labels = atoi(argv[5]);
  
  float *X_train = mallocUni(m*(n-1)), *X_test = mallocUni(t*(n-1));
  int   *y_train = mallocY(m), *y_test = mallocY(t), *y_pred = mallocY(t), *result = mallocUni_int(t*k);
  float *normX_train = mallocUni(m), *normX_test = mallocUni(t);
  float *trainTtest = mallocUni(t*m);

  char **data_train = mallocData(m, n, MAX_LEN);
  char **data_test = mallocData(t, n, MAX_LEN);
  read_csv(m, n, data_train, argv[6]);
  read_csv(t, n, data_test, argv[7]);
  getXandY(m, n, data_train, X_train, y_train);
  getXandY(t, n, data_test, X_test, y_test);
  freeData(m, n, data_train);
  freeData(t, n, data_test);

  printf("Start predicting...\n");
  hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start);
  getSquaredNorm<<<(int)ceil((m*1.0)/BLOCK_SIZE), BLOCK_SIZE>>>(m, n-1, X_train, normX_train);
  getSquaredNorm<<<(int)ceil((m*1.0)/BLOCK_SIZE), BLOCK_SIZE>>>(t, n-1, X_test, normX_test);
  matmulTrainTest(t, m, n-1, X_test, X_train, trainTtest);
  calculateDistance<<<(int)ceil((m*t*1.0)/BLOCK_SIZE), BLOCK_SIZE>>>(m, t, normX_train, normX_test, trainTtest);
  parallel_sort<<<(int)ceil((t*1.0)/BLOCK_SIZE), BLOCK_SIZE>>>(t, m, k, trainTtest, result);
  hipDeviceSynchronize();
  for (int i=0; i<t; i++)
  {
    int *pred_labels = (int *)malloc(k*sizeof(int));
    for (int j=0; j<k; j++) pred_labels[j] = (int)y_train[result[i*k+j]];
    y_pred[i] = major_num(k, labels, pred_labels);
    free(pred_labels);
  }
  hipEventRecord(stop); hipEventSynchronize(stop); hipDeviceSynchronize();
  float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
  printf("Done in %.6f s\n", milliseconds*1e-3);
  float acc = accuracy(t, y_test, y_pred);
  printf("Accuracy: %.6f %%\n", acc);

  freeUni(X_train); freeUni(X_test);
  freeY(y_train); freeY(y_test); freeY(y_pred); freeUni_int(result);
  freeUni(normX_train); freeUni(normX_test);
  freeUni(trainTtest);

  return 0;
}
