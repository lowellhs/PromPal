#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hipblas.h"
#include "helper.c"

#define MAX_LEN 1000

__global__ void getSquaredNorm(int m, int n, float *X, float *normX)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m)
  {
    float norm = 0.0;
    for (int j=0; j<n; j++)
    {
      float val = X[idx*n+j];
      norm += (val*val);
    }
    normX[idx] = norm;
  }
}

__global__ void calculateDistance(int m, int t, float *normX, float *normY, float *xTy)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m*t)
  {
    int i = idx/m, j = idx%m;
    float squaredDist = normX[j] + normY[i] + xTy[idx];
    float dist = sqrt(squaredDist);
    xTy[idx] = dist;
  }
}

__global__ void parallel_sort(int t, int m, int k, float *matrix, int *result)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < t)
  {
    for (int i=0; i<k; i++)
    {
      int min_idx = -1, min_idx_g = -1;
      float min_val;
      for (int j=0; j<m; j++)
      {
        int valid = 1;
        for (int jj=0; jj<i; jj++) valid = valid && (j != result[idx*k+jj]);
        if (valid)
        {
          float val = matrix[idx*m+j];
          if ((min_idx == -1) || (val < min_val))
          {
            min_val = val;
            min_idx = idx*m+j;
            min_idx_g = j;
          }
        }
      }
      result[idx*k+i] = min_idx_g;
    }
  }
}

void matmulTrainTest(int m, int n, int k, float *A, float *B, float *C)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float al = -2.0f, bet = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &al, B, k, A, k, &bet, C, n);
  hipblasDestroy(handle); 
}

float accuracy(int n, int *y1, int *y2) {
  int equals = 0;
  for (int i=0; i<n; i++) {
    if (y1[i] == y2[i]) {
      equals += 1;
    }
  }
  return (equals*1.0/n * 100);
}

int main(int argc, char **argv)
{
  int m = 60000;
  int t = 10000;
  int n = 33;
  int k = 5;
  int labels = 10;
  // int m = 150;
  // int t = 3;
  // int n = 5;
  // int k = 5;
  // int labels = 3;
  
  float *X_train = mallocUni(m*(n-1)), *X_test = mallocUni(t*(n-1));
  int   *y_train = mallocY(m), *y_test = mallocY(t), *y_pred = mallocY(t), *result = mallocUni_int(t*k);
  float *normX_train = mallocUni(m), *normX_test = mallocUni(t);
  float *trainTtest = mallocUni(t*m);

  char **data_train = mallocData(m, n, MAX_LEN);
  char **data_test = mallocData(t, n, MAX_LEN);
  read_csv(m, n, data_train, "test_input/MNIST_train_60k.csv");
  read_csv(t, n, data_test, "test_input/MNIST_test_10k.csv");
  // read_csv(m, n, data_train, "test_input/Iris_150.csv");
  // read_csv(t, n, data_test, "test_input/Iris_test.csv");
  getXandY(m, n, data_train, X_train, y_train);
  getXandY(t, n, data_test, X_test, y_test);
  freeData(m, n, data_train);
  freeData(t, n, data_test);

  // printf("Start predicting...\n");
  hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start);
  getSquaredNorm<<<(int)ceil(m/1024.0), 1024>>>(m, n-1, X_train, normX_train);
  getSquaredNorm<<<(int)ceil(m/1024.0), 1024>>>(t, n-1, X_test, normX_test);
  matmulTrainTest(t, m, n-1, X_test, X_train, trainTtest);
  calculateDistance<<<(int)ceil(m/1024.0), 1024>>>(m, t, normX_train, normX_test, trainTtest);
  parallel_sort<<<(int)ceil((t)/1024.0), 1024>>>(t, m, k, trainTtest, result);
  hipDeviceSynchronize();
  for (int i=0; i<t; i++)
  {
    int *pred_labels = (int *)malloc(k*sizeof(int));
    for (int j=0; j<k; j++) pred_labels[j] = (int)y_train[result[i*k+j]];
    y_pred[i] = major_num(k, labels, pred_labels);
    free(pred_labels);
  }
  print_vector_int(t, y_pred);
  hipEventRecord(stop); hipEventSynchronize(stop); hipDeviceSynchronize();
  float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
  // printf("%.6f\n", milliseconds*1e-3);
  float acc = accuracy(t, y_test, y_pred);
  // printf("Acc: %.6f\n", acc);

  freeUni(X_train); freeUni(X_test);
  freeY(y_train); freeY(y_test); freeY(y_pred); freeUni_int(result);
  freeUni(normX_train); freeUni(normX_test);
  freeUni(trainTtest);

  return 0;
}
