#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "helper.c"

#define MAX_LEN 1000

__global__ void getSquaredNorm(int m, int n, float *X, float *normX)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m*n)
  {
    float norm = 0.0;
    for (int j=0; j<n; j++)
    {
      float val = X[idx*n+j];
      norm += (val*val);
    }
    normX[idx] = norm;
  }
}

void getTopKRows(int m, int k, float *distances, int *indices)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int idxMin, idxMax;
  float maxVal;
  hipblasIsamax(handle, m, distances, 1, &idxMax);
  hipDeviceSynchronize();
  maxVal = distances[idxMax-1];
  for (int i=0; i<k; i++)
  {
    hipblasIsamin(handle, m, distances, 1, &idxMin);
    hipDeviceSynchronize();
    indices[i] = idxMin-1;
    distances[idxMin-1] = maxVal;
  }

  hipblasDestroy(handle);
}

int main(int argc, char **argv)
{
  int m = 150;
  int t = 3;
  int n = 5;
  
  float *X_train = mallocUniY(m*(n-1)), *X_test = mallocUniY(t*(n-1));
  float *y_train = mallocUniY(m), *y_test = mallocUniY(t), *distances = mallocUniY(m);

  char **data_train = mallocData(m, n, MAX_LEN);
  char **data_test = mallocData(t, n, MAX_LEN);
  read_csv(m, n, data_train, "test_input/Iris_150.csv");
  read_csv(t, n, data_test, "test_input/Iris_test.csv");
  getXandY(m, n, data_train, X_train, y_train);
  getXandY(t, n, data_test, X_test, y_test);
  for(int i=0; i<t; i++)
  {
    for (int j=0; j<(n-1); j++)
    {
      printf("%.6f ", X_test[i*(n-1)+j]);
    }
    printf("\n");
  }
  printf("\n");
  for(int i=0; i<t; i++)
  {
    printf("%.6f\n", y_train[i]);
  }
  freeData(m, n, data_train);
  freeData(t, n, data_test);

  freeUniY(X_train); freeUniY(X_test);
  freeUniY(y_train); freeUniY(y_test);

  /*
  int m = atoi(argv[1]);
  int n = atoi(argv[2]);
  int k = atoi(argv[3]);

  float *X_train, *X_test, *distances;
  hipMallocManaged((void **)&X_train, m*n*sizeof(float));
  hipMallocManaged((void **)&X_test, m*n*sizeof(float));
  hipMallocManaged((void **)&X_test, m*sizeof(float));
  for (int i=0; i<m; i++)
  {
    for (int j=0; j<n; j++)
    {
      X_train[i*n+j] = i*n+j;
      X_test[i*n+j] = i*n+j+1;
    }
  }

  hipFree(X_train); hipFree(X_test);
  */
  return 0;
}
