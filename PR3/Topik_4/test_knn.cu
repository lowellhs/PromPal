#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hipblas.h"
#include "helper.c"

#define MAX_LEN 1000

__global__ void getSquaredNorm(int m, int n, float *X, float *normX)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m)
  {
    float norm = 0.0;
    for (int j=0; j<n; j++)
    {
      float val = X[idx*n+j];
      norm += (val*val);
    }
    normX[idx] = norm;
  }
}

__global__ void calculateDistance(int m, int t, float *normX, float *normY, float *xTy)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m*t)
  {
    int i = idx/t, j = idx%t;
    float squaredDist = normX[i] + normY[j] + xTy[idx];
    float dist = sqrt(squaredDist);
    xTy[idx] = dist;
  }
}

void getTopKRows(int m, int k, float *distances, int *indices)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int idxMin, idxMax;
  float maxVal;
  hipblasIsamax(handle, m, distances, 1, &idxMax);
  hipDeviceSynchronize();
  maxVal = distances[idxMax-1];
  for (int i=0; i<k; i++)
  {
    hipblasIsamin(handle, m, distances, 1, &idxMin);
    hipDeviceSynchronize();
    indices[i] = idxMin-1;
    distances[idxMin-1] = maxVal;
  }

  hipblasDestroy(handle);
}

void matmulTrainTest(int m, int n, int k, float *A, float *B, float *C)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float al = -2.0f, bet = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &al, B, k, A, k, &bet, C, n);
  hipblasDestroy(handle); 
}

int main(int argc, char **argv)
{
  int m = 150;
  int t = 3;
  int n = 5;
  
  float *X_train = mallocUni(m*(n-1)), *X_test = mallocUni(t*(n-1));
  float *y_train = mallocUni(m), *y_test = mallocUni(t);
  float *distances = mallocUni(m);
  float *normX_train = mallocUni(m), *normX_test = mallocUni(t);
  float *trainTtest = mallocUni(m*t);

  char **data_train = mallocData(m, n, MAX_LEN);
  char **data_test = mallocData(t, n, MAX_LEN);
  read_csv(m, n, data_train, "test_input/Iris_150.csv");
  read_csv(t, n, data_test, "test_input/Iris_test.csv");
  getXandY(m, n, data_train, X_train, y_train);
  getXandY(t, n, data_test, X_test, y_test);
  freeData(m, n, data_train);
  freeData(t, n, data_test);

  getSquaredNorm<<<(int)ceil(m/1024.0), 1024>>>(m, n-1, X_train, normX_train);
  getSquaredNorm<<<(int)ceil(m/1024.0), 1024>>>(t, n-1, X_test, normX_test);
  matmulTrainTest(m, t, n-1, X_train, X_test, trainTtest);
  calculateDistance<<<(int)ceil(m/1024.0), 1024>>>(m, t, normX_train, normX_test, trainTtest);
  hipDeviceSynchronize();
  print_matrix(m, t, trainTtest);

  print_vector(t, normX_test);
  freeUni(X_train); freeUni(X_test);
  freeUni(y_train); freeUni(y_test);

  /*
  int m = atoi(argv[1]);
  int n = atoi(argv[2]);
  int k = atoi(argv[3]);

  float *X_train, *X_test, *distances;
  hipMallocManaged((void **)&X_train, m*n*sizeof(float));
  hipMallocManaged((void **)&X_test, m*n*sizeof(float));
  hipMallocManaged((void **)&X_test, m*sizeof(float));
  for (int i=0; i<m; i++)
  {
    for (int j=0; j<n; j++)
    {
      X_train[i*n+j] = i*n+j;
      X_test[i*n+j] = i*n+j+1;
    }
  }

  hipFree(X_train); hipFree(X_test);
  */
  return 0;
}
