#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hipblas.h"
#include "helper.c"

#define MAX_LEN 1000

__global__ void getSquaredNorm(int m, int n, float *X, float *normX)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m)
  {
    float norm = 0.0;
    for (int j=0; j<n; j++)
    {
      float val = X[idx*n+j];
      norm += (val*val);
    }
    normX[idx] = norm;
  }
}

__global__ void calculateDistance(int m, int t, float *normX, float *normY, float *xTy)
{
  int idx = blockIdx.x*gridDim.x + threadIdx.x;
  if (idx < m*t)
  {
    int i = idx/m, j = idx%m;
    float squaredDist = normX[j] + normY[i] + xTy[idx];
    float dist = sqrt(squaredDist);
    xTy[idx] = dist;
  }
}

void getTopKRows(int m, int k, float *distances, int *indices)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int idxMin, idxMax;
  float maxVal;
  hipblasIsamax(handle, m, distances, 1, &idxMax);
  hipDeviceSynchronize();
  maxVal = distances[idxMax-1];
  for (int i=0; i<k; i++)
  {
    hipblasIsamin(handle, m, distances, 1, &idxMin);
    hipDeviceSynchronize();
    indices[i] = idxMin-1;
    distances[idxMin-1] = maxVal;
  }

  hipblasDestroy(handle);
}

void matmulTrainTest(int m, int n, int k, float *A, float *B, float *C)
{
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float al = -2.0f, bet = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &al, B, k, A, k, &bet, C, n);
  hipblasDestroy(handle); 
}

int main(int argc, char **argv)
{
  int m = 60000;
  int t = 10000;
  int n = 33;
  int k = 5;
  int labels = 10;
  
  float *X_train = mallocUni(m*(n-1)), *X_test = mallocUni(t*(n-1));
  int   *y_train = mallocY(m), *y_test = mallocY(t), *y_pred = mallocY(t);
  float *normX_train = mallocUni(m), *normX_test = mallocUni(t);
  float *trainTtest = mallocUni(t*m);

  char **data_train = mallocData(m, n, MAX_LEN);
  char **data_test = mallocData(t, n, MAX_LEN);
  read_csv(m, n, data_train, "test_input/MNIST_train_60k.csv");
  read_csv(t, n, data_test, "test_input/MNIST_test_10k.csv");
  getXandY(m, n, data_train, X_train, y_train);
  getXandY(t, n, data_test, X_test, y_test);
  freeData(m, n, data_train);
  freeData(t, n, data_test);

  printf("Start predicting...\n");
  hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start);
  getSquaredNorm<<<(int)ceil(m/1024.0), 1024>>>(m, n-1, X_train, normX_train);
  getSquaredNorm<<<(int)ceil(m/1024.0), 1024>>>(t, n-1, X_test, normX_test);
  matmulTrainTest(t, m, n-1, X_test, X_train, trainTtest);
  calculateDistance<<<(int)ceil(m/1024.0), 1024>>>(m, t, normX_train, normX_test, trainTtest);
  hipEventRecord(stop); hipEventSynchronize(stop); hipDeviceSynchronize();
  float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
  hipDeviceSynchronize();
  for (int i=0; i<t; i++)
  {
    int *indices = (int *)malloc(k*sizeof(int));
    int *pred_labels = (int *)malloc(k*sizeof(int));
    getTopKRows(m, k, &(trainTtest[i*m]), indices);
    for (int j=0; j<k; j++) pred_labels[j] = (int)y_train[indices[j]];
    y_pred[i] = major_num(k, labels, pred_labels);
    free(indices); free(pred_labels);
  }
  // print_vector_int(t, y_pred);
  // hipEventRecord(stop); hipEventSynchronize(stop); hipDeviceSynchronize();
  // float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
  printf("%.6f\n", milliseconds*1e-3);

  freeUni(X_train); freeUni(X_test);
  freeY(y_train); freeY(y_test); freeY(y_pred);
  freeUni(normX_train); freeUni(normX_test);
  freeUni(trainTtest);

  return 0;
}
