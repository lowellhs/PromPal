#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hipblas.h"
#include "helper.c"

#define MAX_LEN 1000

__global__ void parallel_sort(int t, int m, int k, float *matrix, int *result)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < t)
  {
    for (int i=0; i<k; i++)
    {
      int min_idx = -1, min_idx_g = -1;
      float min_val;
      for (int j=0; j<m; j++)
      {
        int valid = 1;
        for (int jj=0; jj<i; jj++) valid = valid && (j != result[idx*k+jj]);
        if (valid)
        {
          float val = matrix[idx*m+j];
          if ((min_idx == -1) || (val < min_val))
          {
            min_val = val;
            min_idx = idx*m+j;
            min_idx_g = j;
          }
        }
      }
      result[idx*k+i] = min_idx_g;
    }
  }
}

int main(int argc, char **argv)
{
  srand(time(NULL));
  int m = atoi(argv[1]), t = atoi(argv[2]), k = atoi(argv[3]);
  float *matrix = mallocUni(t*m);
  int *result = mallocUni_int(t*k);
  for (int i=0; i<t; i++)
  {
    for (int j=0; j<m; j++)
    {
      matrix[i*m+j] = m-j;
    }
  }
  printf("Begin...\n");
  hipDeviceSynchronize();
  parallel_sort<<<(int)ceil((t)/1024.0), 1024>>>(t, m, k, matrix, result);
  hipDeviceSynchronize();
  printf("Done!\n");
  // print_matrix(t, m, matrix);
  print_matrix_int(t, k, result);
  freeUni(matrix);
  freeUni_int(result);
  return 0;
}
