#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "matrixTools.c"
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define TOL 1e-6
#define limit_iter 10000
#define BLOCK_SIZE 50

__device__ int flag;

__global__ void getDLU(int n, float *A, float *D_inv, float *LU)
{
  int i = blockIdx.y*blockDim.y + threadIdx.y;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<n && j<n)
  {
    if (i == j)
    {
      D_inv[i*n+j] = 1.0/(A[i*n+j]);
    }
    else
    {
      LU[i*n+j] = -A[i*n+j];
    }
  }
}

__global__ void checkConvergence(int n, float *x_iter, float *x_iter_new)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<n)
  {
    if (fabs(x_iter[i]-x_iter_new[i]) > TOL) flag = 0;
    x_iter[i] = x_iter_new[i];
  }
}

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
      hipEvent_t start, stop;

      float *A, *b, *x, *x_iter, *x_iter_new, *D_inv, *LU, *T, *c;
      float al, bet;

      int n = atoi(argv[counter]);
      hipMallocManaged(&A, n*n*sizeof(float));
      hipMallocManaged(&b, n*sizeof(float));
      hipMallocManaged(&x, n*sizeof(float));
      hipMallocManaged(&x_iter, n*sizeof(float));
      hipMallocManaged(&x_iter_new, n*sizeof(float));
      hipMallocManaged(&D_inv, n*n*sizeof(float));
      hipMallocManaged(&LU, n*n*sizeof(float));
      hipMallocManaged(&T, n*n*sizeof(float));
      hipMallocManaged(&c, n*sizeof(float));
      
      char file_A[80], file_b[80], file_x[80];
      sprintf(file_A, "test_input/matrix_A_%dx%d.txt", n, n);
      sprintf(file_b, "test_input/vector_b_%dx1.txt", n);
      sprintf(file_x, "test_input/vector_x_%dx1.txt", n);
      read_matrix(n, A, file_A, " ");
      read_vector(n, b, file_b, " ");
      read_vector(n, x, file_x, " ");
      for (int i=0; i<n; i++) x_iter[i] = 0.0;
    
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);

      int blockDim = 32;
      int gridDim = (int)ceil(n/32.0);
      dim3 aa = dim3(gridDim,gridDim);
      dim3 bb = dim3(blockDim,blockDim);
      getDLU<<<aa,bb>>>(n, A, D_inv, LU);
      hipDeviceSynchronize();

      int gridSize = (int)ceil(1.0*n/BLOCK_SIZE);
      hipblasHandle_t handle;
      hipblasCreate(&handle);
      al = 1.0f, bet = 0.0f;
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &al, LU, n, D_inv, n, &bet, T, n);
      hipblasSgemv(handle, HIPBLAS_OP_N, n, n, &al, D_inv, n, b, 1, &bet, c, 1);
      hipDeviceSynchronize();
      int k = 0, isConverged;
      do {
        isConverged = 1;

        hipMemcpyToSymbol(HIP_SYMBOL(flag), &isConverged, sizeof(int));
        hipblasSgemv(handle, HIPBLAS_OP_T, n, n, &al, T, n, x_iter, 1, &bet, x_iter_new, 1);
        hipblasSaxpy(handle, n, &al, c, 1, x_iter_new, 1);
        checkConvergence<<<gridSize, BLOCK_SIZE>>>(n, x_iter, x_iter_new);
        hipblasScopy(handle, n, x_iter_new, 1, x_iter, 1);
        hipMemcpyFromSymbol(&isConverged, HIP_SYMBOL(flag), sizeof(int));
        hipDeviceSynchronize();

        k++;
      } while (k < limit_iter && !isConverged);
      hipblasDestroy(handle);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
      hipDeviceSynchronize();

      // hipError_t code=hipGetLastError();
      // printf("%s\n", hipGetErrorString(code));
      printf("%d ", n);
      printf("%.6f ", milliseconds*1e-3);
      printf("%.9f ", norm_vector(n, x_iter, x));
      printf("%d\n", k);

      hipFree(A); hipFree(b); hipFree(x); hipFree(x_iter); hipFree(x_iter_new); hipFree(D_inv); hipFree(LU);
      hipFree(T); hipFree(c);
    }
  }
  return 0;
}
