#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "matrixTools.c"
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define TOL 1e-6
#define limit_iter 10000

__device__ int flag;

__global__ void jacobiOnDevice(int n, float *A, float *b, float *x_iter, float *x_iter_new)
{
  float sigma = 0;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
  {
    for (int j=0; j<n; j++) {
      if (j != i) {
        sigma = sigma + A[i*n+j]*x_iter[j];
      }
    }
    x_iter_new[i] = (b[i]-sigma)/A[i*n+i];
  }
}

__global__ void checkConvergence(int n, float *x_iter, float *x_iter_new)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<n)
  {
    if (fabs(x_iter[i]-x_iter_new[i]) > TOL) flag = 0;
    x_iter[i] = x_iter_new[i];
  }
}


int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
      hipEvent_t start, stop;
      
      float *A, *b, *x, *x_iter, *x_iter_new;

      int n = atoi(argv[counter]);
      hipMallocManaged(&A, n*n*sizeof(float));
      hipMallocManaged(&b, n*sizeof(float));
      hipMallocManaged(&x, n*sizeof(float));
      hipMallocManaged(&x_iter, n*sizeof(float));
      hipMallocManaged(&x_iter_new, n*sizeof(float));
      
      char file_A[80], file_b[80], file_x[80];
      sprintf(file_A, "test_input/matrix_A_%dx%d.txt", n, n);
      sprintf(file_b, "test_input/vector_b_%dx1.txt", n);
      sprintf(file_x, "test_input/vector_x_%dx1.txt", n);
      read_matrix(n, A, file_A, " ");
      read_vector(n, b, file_b, " ");
      read_vector(n, x, file_x, " ");
      for (int i=0; i<n; i++) x_iter[i] = 0.0;
    
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);
      int k = 0, isConverged;
      do {
        isConverged = 1;

        hipMemcpyToSymbol(HIP_SYMBOL(flag), &isConverged, sizeof(int));
        jacobiOnDevice<<<dim3((int)ceil(n/1024.0),1,1), dim3(1024,1,1)>>>(n, A, b, x_iter, x_iter_new);
        checkConvergence<<<dim3((int)ceil(n/1024.0),1,1), dim3(1024,1,1)>>>(n, x_iter, x_iter_new);
        hipMemcpyFromSymbol(&isConverged, HIP_SYMBOL(flag), sizeof(int));
        hipDeviceSynchronize();

        k++;
      } while (k < limit_iter && !isConverged);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
      hipDeviceSynchronize();

      // hipError_t code=hipGetLastError();
      // printf("%s\n", hipGetErrorString(code));
      printf("%d ", n);
      printf("%.6f ", milliseconds*1e-3);
      printf("%.9f ", norm_vector(n, x_iter, x));
      printf("%d\n", k);
      hipFree(A); hipFree(b); hipFree(x); hipFree(x_iter); hipFree(x_iter_new);
    }
  }
  return 0;
}
