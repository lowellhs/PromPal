#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "matrixTools.c"
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define TOL 1e-6
#define limit_iter 10000

__device__ int flag;

__global__ void jacobiOnDevice(int n, float *A, float *b, float *x_iter)
{
  float sigma = 0, newValue;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
  {
    for (int j=0; j<n; j++) {
      if (j != i) {
        sigma = sigma + A[i*n+j]*x_iter[j];
      }
    }
    newValue = (b[i]-sigma)/A[i*n+i];
    __syncthreads();
    if (fabs(x_iter[i]-newValue) > TOL) flag = 0;
    x_iter[i] = newValue;
  }
}

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
      hipEvent_t start, stop;
      
      float *A, *b, *x, *x_iter;
      float *A_d, *b_d, *x_iter_d;

      int n = atoi(argv[counter]);
      A = (float *)malloc(sizeof(float)*n*n);
      b = (float *)malloc(sizeof(float)*n);
      x = (float *)malloc(sizeof(float)*n);
      x_iter = (float *)malloc(sizeof(float)*n);

      hipMalloc((void **) &A_d, sizeof(float)*n*n);
      hipMalloc((void **) &b_d, sizeof(float)*n);
      hipMalloc((void **) &x_iter_d, sizeof(float)*n);
      
      char file_A[80], file_b[80], file_x[80];
      sprintf(file_A, "test_input/matrix_A_%dx%d.txt", n, n);
      sprintf(file_b, "test_input/vector_b_%dx1.txt", n);
      sprintf(file_x, "test_input/vector_x_%dx1.txt", n);
      read_matrix(n, A, file_A, " ");
      read_vector(n, b, file_b, " ");
      read_vector(n, x, file_x, " ");
      for (int i=0; i<n; i++) x_iter[i] = 0.0;
    
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);

      hipMemcpy(A_d, A, n*n*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(b_d, b, n*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(x_iter_d, x_iter, n*sizeof(float), hipMemcpyHostToDevice);
      hipDeviceSynchronize();


      int k = 0, isConverged;
      do {
        isConverged = 1;

        hipMemcpyToSymbol(HIP_SYMBOL(flag), &isConverged, sizeof(int));
        jacobiOnDevice<<<dim3((int)ceil(n/1024.0),1,1),dim3(1024,1,1)>>>(n, A_d, b_d, x_iter_d);
        hipMemcpyFromSymbol(&isConverged, HIP_SYMBOL(flag), sizeof(int));
        hipDeviceSynchronize();

        k++;
      } while (k < limit_iter && !isConverged);

      hipMemcpy(x_iter, x_iter_d, n*sizeof(float), hipMemcpyDeviceToHost);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
      hipDeviceSynchronize();

      printf("%d ", n);
      printf("%.6f ", milliseconds*1e-3);
      printf("%.9f ", norm_vector(n, x_iter, x));
      printf("%d\n", k);
      free(A); free(b); free(x); free(x_iter);
      hipFree(A_d); hipFree(b_d); hipFree(x_iter_d);
    }
  }
  return 0;
}
