#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "matrixTools.c"
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define TOL 1e-6
#define limit_iter 10000

__global__ void jacobiOnDevice(int n, float *A, float *b, float *x_iter, float *x_iter_new)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < n)
  {
    x_iter_new[idx] = b[idx];
    for (int j=0; j<n; j++) {
      if (j != idx) {
        x_iter_new[idx] -= (A[idx*n+j] * x_iter[j]);
      }
    }
    x_iter_new[idx] /= A[idx*n+idx];
    x_iter[idx] = x_iter_new[idx];
  }
}

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
      hipEvent_t start, stop;
      
      float *A, *b, *x, *x_iter, *x_iter_new;
      float *A_d, *b_d, *x_iter_d, *x_iter_new_d;
      float dist;

      int n = atoi(argv[counter]);
      A = (float *)malloc(sizeof(float)*n*n);
      b = (float *)malloc(sizeof(float)*n);
      x = (float *)malloc(sizeof(float)*n);
      x_iter = (float *)malloc(sizeof(float)*n);
      x_iter_new = (float *)malloc(sizeof(float)*n);
      hipMalloc((void **) &A_d, sizeof(float)*n*n);
      hipMalloc((void **) &b_d, sizeof(float)*n);
      hipMalloc((void **) &x_iter_d, sizeof(float)*n);
      hipMalloc((void **) &x_iter_new_d, sizeof(float)*n);
      
      char file_A[80], file_b[80], file_x[80];
      sprintf(file_A, "test_input/matrix_A_%dx%d.txt", n, n);
      sprintf(file_b, "test_input/vector_b_%dx1.txt", n);
      sprintf(file_x, "test_input/vector_x_%dx1.txt", n);
      read_matrix(n, A, file_A, " ");
      read_vector(n, b, file_b, " ");
      read_vector(n, x, file_x, " ");
      for (int i=0; i<n; i++) x_iter[i] = 0.0;
    
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);

      hipMemcpy(A_d, A, n*n*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(b_d, b, n*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(x_iter_d, x_iter, n*sizeof(float), hipMemcpyHostToDevice);


      int k = 0;
      do {
        k++;

        jacobiOnDevice<<<(int)ceil(n/1024.0),1024>>>(n, A_d, b_d, x_iter_d, x_iter_new_d);
        hipDeviceSynchronize();
        hipMemcpy(x_iter_new, x_iter_new_d, n*sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        dist = norm_vector(n, x_iter_new, x_iter);
        for (int i=0; i<n; i++) x_iter[i] = x_iter_new[i];
      } while (k < limit_iter && dist > TOL);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
      hipDeviceSynchronize();

      printf("%d ", n);
      printf("%.6f ", milliseconds*1e-3);
      printf("%.9f ", norm_vector(n, x_iter, x));
      printf("%d\n", k);
      free(A); free(b); free(x); free(x_iter); free(x_iter_new);
      hipFree(A_d); hipFree(b_d); hipFree(x_iter); hipFree(x_iter_new);
    }
  }
  return 0;
}
