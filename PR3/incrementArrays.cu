// incrementArray.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

struct timeval t1, t2;

void incrementArrayOnHost(float *a, int N)
{
  int i;
  for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx]+1.f;
}

int main(void)
{
  float *a_h, *b_h; // pointers to host memory
  float *a_d; // pointer to device memory
  int i, N = 1e+9;
  size_t size = N*sizeof(float);

  // allocate arrays on host
  a_h = (float *)malloc(size);
  b_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &a_d, size);
  // initialization of host data
  for (i=0; i<N; i++) a_h[i] = (float)i;
  // copy data from host to device
  hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
  
  
  gettimeofday(&t1, 0);
  // do calculation on host
  incrementArrayOnHost(a_h, N);
  gettimeofday(&t2, 0);
  double time1 = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
  printf("Time to generate:  %3.1f ms \n", time1);

  gettimeofday(&t1, 0);
  // do calculation on device:
  // Part 1 of 2. Compute execution configuration
  int blockSize = 8;
  int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
  // Part 2 of 2. Call incrementArrayOnDevice kernel
  incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
  // Retrieve result from device and store in b_h
  hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  gettimeofday(&t2, 0);
  double time2 = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
  printf("Time to generate:  %3.1f ms \n", time2);

  // check results
  for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);
  // cleanup
  free(a_h); free(b_h); hipFree(a_d);
}
