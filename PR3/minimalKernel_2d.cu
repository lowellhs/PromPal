#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void kernelId(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = id;
}

__global__ void kernelBlockIdxX(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = blockIdx.x;
}

__global__ void kernelBlockIdxY(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = blockIdx.y;
}

__global__ void kernelThreadIdxX(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = threadIdx.x;
}

__global__ void kernelThreadIdxY(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = threadIdx.y;
}

void printArr(int *a, int N)
{
  for (int i=0; i<N; i++) printf("%2d ", a[i]);
  printf("\n");
}

int main(void)
{
  for (int kk=0; kk<5; kk++)
  {
    int *a_h, *b_h; //pointers to host memory
    int *a_d; //pointers to device memory
    int i;
    int N = 24;
    size_t size = N*sizeof(int);
    
    //allocate array on host
    a_h = (int *)malloc(size);
    b_h = (int *)malloc(size);

    //allocate array on device
    hipMalloc((void **) &a_d, size);
    
    //initialization of host data
    for (i=0; i<N; i++) a_h[i] = 0;

    //copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(int)*N, hipMemcpyHostToDevice);
    
    //do calculation on host
    dim3 dimGrid = dim3(2,2);
    dim3 dimBlock = dim3(3,2);
    if (kk==0) kernelId<<<dimGrid,dimBlock>>>(a_d);
    if (kk==1) kernelBlockIdxX<<<dimGrid,dimBlock>>>(a_d);
    if (kk==2) kernelBlockIdxY<<<dimGrid,dimBlock>>>(a_d);
    if (kk==3) kernelThreadIdxX<<<dimGrid,dimBlock>>>(a_d);
    if (kk==4) kernelThreadIdxY<<<dimGrid,dimBlock>>>(a_d);

    //retrieve result from device and store in b_h
    hipMemcpy(b_h, a_d, sizeof(int)*N, hipMemcpyDeviceToHost);
    
    //print out the result
    printArr(b_h, N);

    //cleanup
    free(a_h); free(b_h); hipFree(a_d);
  }
}
