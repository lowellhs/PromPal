#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void kernel(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = 7;
}

int main(void)
{
  int *a_h, *b_h; //pointers to host memory
  int *a_d; //pointers to device memory
  int i;
  int N = 10;
  
  //allocate array on host
  a_h = (int *)malloc(sizeof(int));
  b_h = (int *)malloc(sizeof(int));

  //allocate array on device
  hipMalloc((void **) &a_d, sizeof(int));
  
  //initialization of host data
  for (i=0; i<N; i++) a_h[i] = 0;

  //copy data from host to device
  hipMemcpy(a_d, a_h, sizeof(int)*N, hipMemcpyHostToDevice);
  
  //do calculation on host
  kernel<<<1,1>>>(a_h);

  //retrieve result from device and store in b_h
  hipMemcpy(b_h, a_d, sizeof(int)*N, hipMemcpyDeviceToHost);
  
  //print out the result
  for (i=0; i<N; i++) printf("%d", b_h[i]);

  //cleanup
  free(a_h); free(b_h); hipFree(a_d);
}
