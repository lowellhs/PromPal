#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cstdlib>
#include "hipblas.h"

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
    hipEvent_t start, stop;

    float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
    float *A_d, *B_d, *C_d;         // pointers to device memory

    // size of matrix (n*n)
    int n = atoi(argv[counter]);
    size_t size = n*n*sizeof(float);

    // allocate array on host
    A_h  = (float *)malloc(size);
    B_h  = (float *)malloc(size);
    C_h  = (float *)malloc(size);
    C2_h = (float *)malloc(size);

    // allocate array on device
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // initializtion of host data
    initIdentityMatrix(n, A_h);
    initRandomMatrix(n, B_h);
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // copy data from host to device
    hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

    // do calculation on device
    int lda=n, ldb=n, ldc=n;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, B_d, lda, A_d, ldb, beta, C_d, ldc);
    hipblasDestroy(handle);
    hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
    hipDeviceSynchronize();

    float err = errorMatrix(n, C2_h, B_h);
    printf("%d ", n);
    printf("%.6f ", milliseconds*1e-3);
    printf("%.6f\n", err);

    // Cleanup
    free(A_h); free(B_h); free(C_h); free(C2_h);
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
    }
  }
}
