#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <cstdlib>
#include "hipblas.h"

int main(int argc, char **argv)
{
  struct timeval startCPU, stopCPU, startGPU, stopGPU;

  float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
  float *A_d, *B_d, *C_d;         // pointers to device memory

  // size of matrix (n*n)
  int n = atoi(argv[1]);
  size_t size = n*n*sizeof(float);

  // allocate array on host
  A_h  = (float *)malloc(size);
  B_h  = (float *)malloc(size);
  C_h  = (float *)malloc(size);
  C2_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &A_d, size);
  hipMalloc((void **) &B_d, size);
  hipMalloc((void **) &C_d, size);

  // initializtion of host data
  initIdentityMatrix(n, A_h);
  initRandomMatrix(n, B_h);
  
  gettimeofday(&startGPU, 0);
  // copy data from host to device
  hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

  // do calculation on device
  int lda=n, ldb=n, ldc=n;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A_d, lda, B_d, ldb, beta, C_d, ldc);
  hipblasDestroy(handle);
  hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  gettimeofday(&stopGPU, 0);

  // print matrix OR check matrix OR print time
  if (atoi(argv[2]) == 0) printMatrix(n, C2_h);
  if (atoi(argv[2]) == 1) // compare to CPU time and CPU result
  {
    // do calculation on host
    gettimeofday(&startCPU, 0);
    matmul(n, A_h, B_h, C_h);
    float err = errorMatrix(n, C2_h, C_h);
    gettimeofday(&stopCPU, 0);

    printf("CPU time : %.6f\n", (stopCPU.tv_sec+stopCPU.tv_usec*1e-6)-(startCPU.tv_sec+startCPU.tv_usec*1e-6));
    printf("error    : %.6f\n", err);
  }
  if (atoi(argv[2]) == 2) // if I.B = C means B should equals C (A is identity matrix)
  {
    float err = errorMatrix(n, C2_h, B_h);
    printf("error    : %.6f\n", err);
  }
  printf("GPU time : %.6f\n", (stopGPU.tv_sec+stopGPU.tv_usec*1e-6)-(startGPU.tv_sec+startGPU.tv_usec*1e-6));

  // Cleanup
  free(A_h); free(B_h); free(C_h); free(C2_h);
  hipFree(A_d); hipFree(B_d); hipFree(C_d);
}

