#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <cstdlib>
#include "hipblas.h"

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  for (int counter=2; counter < 2+sizeCounter; counter++)
  {
    struct timeval startGPU, stopGPU;

    float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
    float *A_d, *B_d, *C_d;         // pointers to device memory

    // size of matrix (n*n)
    int n = atoi(argv[counter]);
    size_t size = n*n*sizeof(float);

    // allocate array on host
    A_h  = (float *)malloc(size);
    B_h  = (float *)malloc(size);
    C_h  = (float *)malloc(size);
    C2_h = (float *)malloc(size);

    // allocate array on device
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // initializtion of host data
    initIdentityMatrix(n, A_h);
    initRandomMatrix(n, B_h);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    gettimeofday(&startGPU, 0);
    hipEventRecord(start);
    // copy data from host to device
    hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

    // do calculation on device
    int lda=n, ldb=n, ldc=n;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A_d, lda, B_d, ldb, beta, C_d, ldc);
    hipblasDestroy(handle);
    hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipDeviceSynchronize();
    gettimeofday(&stopGPU, 0);

    float err = errorMatrix(n, C2_h, B_h);
    printf("%d ", n);
    //printf("%.6f ", (stopGPU.tv_sec+stopGPU.tv_usec*1e-6)-(startGPU.tv_sec+startGPU.tv_usec*1e-6));
    printf("%.6f ", milliseconds*1e-6);
    printf("%.6f\n", err);

    // Cleanup
    free(A_h); free(B_h); free(C_h); free(C2_h);
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
  }
}
