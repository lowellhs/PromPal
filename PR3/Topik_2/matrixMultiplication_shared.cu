#include "hip/hip_runtime.h"
#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 32

__global__ void matmulOnDevice(int n, float *A, float *B, float *C)
{
  __shared__ float  a[TILE_WIDTH][TILE_WIDTH], b[TILE_WIDTH][TILE_WIDTH];
  int col =  blockIdx.x*blockDim.x + threadIdx.x;
  int row =  blockIdx.y*blockDim.y + threadIdx.y;

  float c = 0.0f;
  for (int k=0; k < (TILE_WIDTH+n-1)/TILE_WIDTH; k++)
  {
    if (k*TILE_WIDTH + threadIdx.x < n && row < n)
      a[threadIdx.y][threadIdx.x] = A[row*n+k*TILE_WIDTH+threadIdx.x];
    else
      a[threadIdx.y][threadIdx.x] = 0.0;

    if (k*TILE_WIDTH + threadIdx.y < n && col < n)
      b[threadIdx.y][threadIdx.x] = B[col+n*(k*TILE_WIDTH+threadIdx.y)];
    else
      b[threadIdx.y][threadIdx.x] = 0.0;
    __syncthreads();

    for (int kk=0; kk<TILE_WIDTH; ++kk)
      c += a[threadIdx.y][kk]*b[kk][threadIdx.x]; 
    __syncthreads();
  }

  if (row < n && col < n)
  {
    C[row*n+col] = c;
  }
}

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
    hipEvent_t start, stop;

    float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
    float *A_d, *B_d, *C_d;         // pointers to device memory

    // size of matrix (n*n)
    int n = atoi(argv[counter]);
    size_t size = n*n*sizeof(float);

    // allocate array on host
    A_h  = (float *)malloc(size);
    B_h  = (float *)malloc(size);
    C_h  = (float *)malloc(size);
    C2_h = (float *)malloc(size);

    // allocate array on device
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // initializtion of host data
    initIdentityMatrix(n, A_h);
    initRandomMatrix(n, B_h);
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // copy data from host to device
    hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

    // do calculation on device
    // Part 1 of 2. Compute execution configuration
    dim3 gridDim, blockDim;
    if (argc == (3+sizeCounter+4))
    {
      gridDim = dim3(atoi(argv[argc-4]), atoi(argv[argc-3]));
      blockDim = dim3(atoi(argv[argc-2]), atoi(argv[argc-1]));
    }
    else
    {
      if (n*n <= 1024)
      {
        gridDim = dim3(1, 1);
        blockDim = dim3(n, n);
      }
      else
      {
        gridDim = dim3((int)ceil(n/32.0), (int)ceil(n/32.0));
        blockDim = dim3(32, 32);
      }
    }
    
    // Part 2 of 2. Call matmulOnDevice kernel
    matmulOnDevice<<<gridDim, blockDim>>>(n, A_d, B_d, C_d);
    hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
    hipDeviceSynchronize();

    float err = errorMatrix(n, C2_h, B_h);
    printf("%d %.6f %.6f (%d,%d) (%d,%d)\n", n, milliseconds*1e-3, err, gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    // Cleanup
    free(A_h); free(B_h); free(C_h); free(C2_h);
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
    }
  }
}

