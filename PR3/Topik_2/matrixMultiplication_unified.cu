#include "hip/hip_runtime.h"
#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

__global__ void matmulOnDevice(int n, float *A, float *B, float *C)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  if (idx < n && idy < n)
  {
    float temp = 0.0;
    for (int i=0; i<n; i++)
    {
      temp += A[n*idy + i]*B[n*i + idx];
    }
    C[n*idy + idx] = temp;
  }
}

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  int tests = atoi(argv[2]);
  for (int counter=3; counter < 3+sizeCounter; counter++)
  {
    for (int testCounter=0; testCounter < tests; testCounter++)
    {
    hipEvent_t start, stop;

    float *A, *B, *C;

    // size of matrix (n*n)
    int n = atoi(argv[counter]);
    size_t size = n*n*sizeof(float);

  	hipMallocManaged(&A,size);
  	hipMallocManaged(&B,size);
  	hipMallocManaged(&C,size);

    // initializtion of host data
    initIdentityMatrix(n, A);
    initRandomMatrix(n, B);
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // do calculation on device
    // Part 1 of 2. Compute execution configuration
    dim3 gridDim, blockDim;
    if (argc == (3+sizeCounter+4))
    {
      gridDim = dim3(atoi(argv[argc-4]), atoi(argv[argc-3]));
      blockDim = dim3(atoi(argv[argc-2]), atoi(argv[argc-1]));
    }
    else
    {
      if (n*n <= 1024)
      {
        gridDim = dim3(1, 1);
        blockDim = dim3(n, n);
      }
      else
      {
        gridDim = dim3((int)ceil(n/32.0), (int)ceil(n/32.0));
        blockDim = dim3(32, 32);
      }
    }
    
    // Part 2 of 2. Call matmulOnDevice kernel
    matmulOnDevice<<<gridDim, blockDim>>>(n, A, B, C);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0; hipEventElapsedTime(&milliseconds, start, stop);
    hipDeviceSynchronize();

    float err = errorMatrix(n, C, B);
    printf("%d (%d,%d) (%d,%d) ", n, gridDim.x, gridDim.y, blockDim.x, blockDim.y);
    printf("%.6f ", milliseconds*1e-3);
    printf("%.6f\n", err);

    // Cleanup
    hipFree(A); hipFree(B); hipFree(C);
    }
  }
}

