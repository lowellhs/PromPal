#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void matmulOnDevice(int n, float *A, float *B, float *C)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  if (idx < n && idy < n)
  {
    float temp = 0.0;
    for (int i=0; i<n; i++)
    {
      temp += A[n*idy + i]*B[n*i + idx];
    }
    C[n*idy + idx] = temp;
  }
}

void matmul(int n, float *A, float *B, float *C)
{
  for (int k=0; k<n; k++)
  {
    for (int j=0; j<n; j++)
    {
      C[n*k+j] = 0;
      for (int i=0; i<n; i++)
      {
        C[n*k+j] = C[n*k+j] + A[n*k+i] * B[n*i+j];
      }
    }
  }
}

void printMatrix(int n, float *A)
{
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      printf("%10.3f ", A[n*i+j]);
    }
    printf("\n");
  }
}

void checkMatrix(int n, float *A, float *B)
{
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      assert(A[n*i+j] == B[n*i+j]);
    }
  }
}

int main(int argc, char **argv)
{
  float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
  float *A_d, *B_d, *C_d;         // pointers to device memory

  // size of matrix (n*n)
  int n = atoi(argv[1]);
  size_t size = n*n*sizeof(float);

  // allocate array on host
  A_h  = (float *)malloc(size);
  B_h  = (float *)malloc(size);
  C_h  = (float *)malloc(size);
  C2_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &A_d, size);
  hipMalloc((void **) &B_d, size);
  hipMalloc((void **) &C_d, size);

  // initializtion of host data
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      A_h[n*i+j] = 0.0;
      B_h[n*i+j] = n*i+j;
      if (i==j) A_h[n*i+j] = 1.0;
    }
  }

  // copy data from host to device
  hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

  // do calculation on device
  // Part 1 of 2. Compute execution configuration
  int gridDimX = atoi(argv[2]);
  int gridDimY = atoi(argv[3]);
  int blockDimX = atoi(argv[4]);
  int blockDimY = atoi(argv[5]);
  dim3 gridSize = dim3(gridDimX, gridDimY);
  dim3 blockSize = dim3(blockDimX, blockDimY);
  printf("blockDim: (%d,%d), gridDim: (%d,%d)\n", blockDimX, blockDimY, gridDimX, gridDimY);

  // Part 2 of 2. Call matmulOnDevice kernel
  matmulOnDevice<<<gridSize, blockSize>>>(n, A_d, B_d, C_d);
  hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

  // print matrix OR check matrix OR print time
  hipDeviceSynchronize();
  if (argc == 7 && atoi(argv[6]) == 0) printMatrix(n, C2_h);
  if (argc == 7 && atoi(argv[6]) == 1)
  {
    // do calculation on host
    matmul(n, A_h, B_h, C_h);
    checkMatrix(n, C2_h, C_h);
  }
  if (argc == 7 && atoi(argv[6]) == 2) printf("time");

  // Cleanup
  free(A_h);
  free(B_h);
  free(C_h);
  free(C2_h);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

}
