#include "hip/hip_runtime.h"
#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

void randomMatrixGenerator(int n, float *A)
{
  srand(time(0));
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++) 
    {
      A[n*i+j] = rand();
    }
  }
}

void initIdentityMatrix(int n, float *I)
{
  for(int i=0; i<n; i++)
  {
    for(int j=0; j<n; j++)
    {
      if(i==j) I[n*i+j] = 1.0;
      else I[n*i+j] = 0.0;
    }
  }
  
}

__global__ void matmulOnDevice(int n, float *A, float *B, float *C)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  if (idx < n && idy < n)
  {
    float temp = 0.0;
    for (int i=0; i<n; i++)
    {
      temp += A[n*idy + i]*B[n*i + idx];
    }
    C[n*idy + idx] = temp;
  }
}

void matmul(int n, float *A, float *B, float *C)
{
  for (int k=0; k<n; k++)
  {
    for (int j=0; j<n; j++)
    {
      C[n*k+j] = 0;
      for (int i=0; i<n; i++)
      {
        C[n*k+j] = C[n*k+j] + A[n*k+i] * B[n*i+j];
      }
    }
  }
}

void printMatrix(int n, float *A)
{
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      printf("%10.3f ", A[n*i+j]);
    }
    printf("\n");
  }
}

float errorMatrix(int n, float *A, float *B)
{
  float res = 0.0;
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      res += (A[n*i+j] - B[n*i+j]);
    }
  }
  return res;
}

int main(int argc, char **argv)
{
  struct timeval startCPU, stopCPU, startGPU, stopGPU;

  float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
  float *A_d, *B_d, *C_d;         // pointers to device memory

  // size of matrix (n*n)
  int n = atoi(argv[1]);
  size_t size = n*n*sizeof(float);

  // allocate array on host
  A_h  = (float *)malloc(size);
  B_h  = (float *)malloc(size);
  C_h  = (float *)malloc(size);
  C2_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &A_d, size);
  hipMalloc((void **) &B_d, size);
  hipMalloc((void **) &C_d, size);

  // initializtion of host data
  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      A_h[n*i+j] = 0.0;
      B_h[n*i+j] = n*i+j;
      if (i==j) A_h[n*i+j] = 1.0;
    }
  }
  
  gettimeofday(&startGPU, 0);
  // copy data from host to device
  hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

  // do calculation on device
  // Part 1 of 2. Compute execution configuration
  int gridDimX = atoi(argv[2]);
  int gridDimY = atoi(argv[3]);
  int blockDimX = atoi(argv[4]);
  int blockDimY = atoi(argv[5]);
  dim3 gridSize = dim3(gridDimX, gridDimY);
  dim3 blockSize = dim3(blockDimX, blockDimY);
  printf("blockDim: (%d,%d), gridDim: (%d,%d)\n", blockDimX, blockDimY, gridDimX, gridDimY);

  // Part 2 of 2. Call matmulOnDevice kernel
  matmulOnDevice<<<gridSize, blockSize>>>(n, A_d, B_d, C_d);
  hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  gettimeofday(&stopGPU, 0);

  // print matrix OR check matrix OR print time
  if (argc == 7 && atoi(argv[6]) == 0) printMatrix(n, C2_h);
  if (argc == 7 && atoi(argv[6]) == 1) // compare to CPU time and CPU result
  {
    // do calculation on host
    gettimeofday(&startCPU, 0);
    matmul(n, A_h, B_h, C_h);
    float err = errorMatrix(n, C2_h, C_h);
    gettimeofday(&stopCPU, 0);

    printf("CPU time : %.6f\n", (stopCPU.tv_sec+stopCPU.tv_usec*1e-6)-(startCPU.tv_sec+startCPU.tv_usec*1e-6));
    printf("error    : %.6f\n", err);
  }
  if (argc == 7 && atoi(argv[6]) == 2) // if I.B = C means B should equals C (A is identity matrix)
  {
    float err = errorMatrix(n, C2_h, B_h);
    printf("error    : %.6f\n", err);
  }
  printf("GPU time : %.6f\n", (stopGPU.tv_sec+stopGPU.tv_usec*1e-6)-(startGPU.tv_sec+startGPU.tv_usec*1e-6));

  // Cleanup
  free(A_h);
  free(B_h);
  free(C_h);
  free(C2_h);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

}

