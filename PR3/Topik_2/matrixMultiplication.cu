#include "hip/hip_runtime.h"
#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

__global__ void matmulOnDevice(int n, float *A, float *B, float *C)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  if (idx < n && idy < n)
  {
    float temp = 0.0;
    for (int i=0; i<n; i++)
    {
      temp += A[n*idy + i]*B[n*i + idx];
    }
    C[n*idy + idx] = temp;
  }
}

int main(int argc, char **argv)
{
  int sizeCounter = atoi(argv[1]);
  for (int counter=2; counter < 2+sizeCounter; counter++)
  {
    struct timeval startGPU, stopGPU;

    float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
    float *A_d, *B_d, *C_d;         // pointers to device memory

    // size of matrix (n*n)
    int n = atoi(argv[counter]);
    size_t size = n*n*sizeof(float);

    // allocate array on host
    A_h  = (float *)malloc(size);
    B_h  = (float *)malloc(size);
    C_h  = (float *)malloc(size);
    C2_h = (float *)malloc(size);

    // allocate array on device
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // initializtion of host data
    initIdentityMatrix(n, A_h);
    initRandomMatrix(n, B_h);
    
    gettimeofday(&startGPU, 0);
    // copy data from host to device
    hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

    // do calculation on device
    // Part 1 of 2. Compute execution configuration
    dim3 gridDim, blockDim;
    if (argc == (2+sizeCounter+4))
    {
      gridDim = dim3(atoi(argv[argc-4]), atoi(argv[argc-3]));
      blockDim = dim3(atoi(argv[argc-2]), atoi(argv[argc-1]));
    }
    else
    {
      if (n*n <= 1024)
      {
        gridDim = dim3(1, 1);
        blockDim = dim3(n, n);
      }
      else
      {
        gridDim = dim3((int)ceil(n/32.0), (int)ceil(n/32.0));
        blockDim = dim3(32, 32);
      }
    }
    
    // Part 2 of 2. Call matmulOnDevice kernel
    matmulOnDevice<<<gridDim, blockDim>>>(n, A_d, B_d, C_d);
    hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    gettimeofday(&stopGPU, 0);

    float err = errorMatrix(n, C2_h, B_h);
    printf("%d (%d,%d) (%d,%d) ", n, gridDim.x, gridDim.y, blockDim.x, blockDim.y);
    printf("%.6f ", (stopGPU.tv_sec+stopGPU.tv_usec*1e-6)-(startGPU.tv_sec+startGPU.tv_usec*1e-6));
    printf("%.6f\n", err);

    // Cleanup
    free(A_h); free(B_h); free(C_h); free(C2_h);
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
  }
}

