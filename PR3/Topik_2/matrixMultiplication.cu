#include "hip/hip_runtime.h"
#include "matrixTools.c"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

__global__ void matmulOnDevice(int n, float *A, float *B, float *C)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  if (idx < n && idy < n)
  {
    float temp = 0.0;
    for (int i=0; i<n; i++)
    {
      temp += A[n*idy + i]*B[n*i + idx];
    }
    C[n*idy + idx] = temp;
  }
}

int main(int argc, char **argv)
{
  struct timeval startCPU, stopCPU, startGPU, stopGPU;

  float *A_h, *B_h, *C_h, *C2_h;  // pointers to host memory
  float *A_d, *B_d, *C_d;         // pointers to device memory

  // size of matrix (n*n)
  int n = atoi(argv[1]);
  size_t size = n*n*sizeof(float);

  // allocate array on host
  A_h  = (float *)malloc(size);
  B_h  = (float *)malloc(size);
  C_h  = (float *)malloc(size);
  C2_h = (float *)malloc(size);

  // allocate array on device
  hipMalloc((void **) &A_d, size);
  hipMalloc((void **) &B_d, size);
  hipMalloc((void **) &C_d, size);

  // initializtion of host data
  initIdentityMatrix(n, A_h);
  initRandomMatrix(n, B_h);
  
  gettimeofday(&startGPU, 0);
  // copy data from host to device
  hipMemcpy(A_d, A_h, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, n*n*sizeof(float), hipMemcpyHostToDevice);

  // do calculation on device
  // Part 1 of 2. Compute execution configuration
  dim3 gridDim, blockDim;
  if (argc == 7)
  {
    gridDim = dim3(atoi(argv[3]), atoi(argv[4]));
    blockDim = dim3(atoi(argv[5]), atoi(argv[6]));
  }
  else
  {
    if (n*n <= 1024)
    {
      gridDim = dim3(1, 1);
      blockDim = dim3(n, n);
    }
    else
    {
      gridDim = dim3((int)ceil(n/32.0), (int)ceil(n/32.0));
      blockDim = dim3(32, 32);
    }
  }
  printf("gridDim: (%d,%d), blockDim: (%d,%d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
  
  // Part 2 of 2. Call matmulOnDevice kernel
  matmulOnDevice<<<gridDim, blockDim>>>(n, A_d, B_d, C_d);
  hipMemcpy(C2_h, C_d, n*n*sizeof(float), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  gettimeofday(&stopGPU, 0);

  // print matrix OR check matrix OR print time
  if (atoi(argv[2]) == 0) printMatrix(n, C2_h);
  if (atoi(argv[2]) == 1) // compare to CPU time and CPU result
  {
    // do calculation on host
    gettimeofday(&startCPU, 0);
    matmul(n, A_h, B_h, C_h);
    float err = errorMatrix(n, C2_h, C_h);
    gettimeofday(&stopCPU, 0);

    printf("CPU time : %.6f\n", (stopCPU.tv_sec+stopCPU.tv_usec*1e-6)-(startCPU.tv_sec+startCPU.tv_usec*1e-6));
    printf("error    : %.6f\n", err);
  }
  if (atoi(argv[2]) == 2) // if I.B = C means B should equals C (A is identity matrix)
  {
    float err = errorMatrix(n, C2_h, B_h);
    printf("error    : %.6f\n", err);
  }
  printf("GPU time : %.6f\n", (stopGPU.tv_sec+stopGPU.tv_usec*1e-6)-(startGPU.tv_sec+startGPU.tv_usec*1e-6));

  // Cleanup
  free(A_h); free(B_h); free(C_h); free(C2_h);
  hipFree(A_d); hipFree(B_d); hipFree(C_d);
}

