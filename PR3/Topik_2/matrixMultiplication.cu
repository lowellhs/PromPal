#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

void matmul(int n, float *A, float *B, float *C)
{
  for (int k=0; k<n; k++) {
    for (int j=0; j<n; j++) {
      C[n*k+j] = 0;
      for (int i=0; i<n; i++) {
        C[n*k+j] = C[n*k+j] + A[n*k+i] * B[n*i+j];
      }
    }
  }
}

int main(int argc, char **argv)
{
  int n = 8;
  size_t size = n*n*sizeof(float);

  float *A = (float *)malloc(size);
  float *B = (float *)malloc(size);
  float *C = (float *)malloc(size);

  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      A[n*i+j] = (float)n*i+j;
      B[n*i+j] = (float)n*i+j;
      C[n*i+j] = 0.0;
    }
  }

  matmul(n, A, B, C);

  for (int i=0; i<n; i++)
  {
    for (int j=0; j<n; j++)
    {
      printf("%10.4f ", C[n*i+j]);
    }
    printf("\n");
  }

  free(A);
  free(B);
  free(C);

}
