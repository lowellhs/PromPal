// incrementArray.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void incrementArrayOnHost(float *a, unsigned long N)
{
  unsigned long i;
  for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, unsigned long N)
{
  unsigned long idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned long idy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned long id  = idy*gridDim.x*blockDim.x + idx;
  if (id<N) a[id] = a[id]+1.f;
}

int main(int argc, char **argv)
{
  float *a_h, *b_h; // pointers to host memory
  float *a_d; // pointer to device memory
  unsigned long i, N = strtoul(argv[1], NULL, 10);
  size_t size = N*sizeof(float);

  // allocate arrays on host
  a_h = (float *)malloc(size);
  b_h = (float *)malloc(size);

  // allocate array on device
  gpuErrchk( hipMalloc((void **) &a_d, size) );
  // initialization of host data
  for (i=0; i<N; i++) a_h[i] = (float)i;
  // copy data from host to device
  gpuErrchk( hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice) );
  
  // do calculation on host
  incrementArrayOnHost(a_h, N);

  // do calculation on device:
  // Part 1 of 2. Compute execution configuration
  unsigned long blockDimX = strtoul(argv[2], NULL, 10);
  unsigned long blockDimY = strtoul(argv[3], NULL, 10);
  unsigned long gridDimX = strtoul(argv[4], NULL, 10);
  unsigned long gridDimY = strtoul(argv[5], NULL, 10);

  dim3 blockSize = dim3(blockDimX, blockDimY);
  dim3 gridSize = dim3(gridDimX, gridDimY);
  printf("blockDim: (%lu,%lu), gridDim: (%lu,%lu)\n", blockDimX, blockDimY, gridDimX, gridDimY);

  // Part 2 of 2. Call incrementArrayOnDevice kernel
  incrementArrayOnDevice <<< gridSize, blockSize >>> (a_d, N);
  // Retrieve result from device and store in b_h
  gpuErrchk( hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost) );

  // cudaError err = cudaGetLastError();
  // if ( cudaSuccess != err ) printf("cudaCheckError() failed: %s\n", cudaGetErrorString(err));
  hipDeviceSynchronize();
  
  // check results
  for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);

  // cleanup
  free(a_h); free(b_h); hipFree(a_d);
}
