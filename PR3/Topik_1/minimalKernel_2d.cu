#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void kernelId(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = id;
}

__global__ void kernelBlockIdxX(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = blockIdx.x;
}

__global__ void kernelBlockIdxY(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = blockIdx.y;
}

__global__ void kernelThreadIdxX(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = threadIdx.x;
}

__global__ void kernelThreadIdxY(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int id = idy*blockDim.x*gridDim.x + idx;
  a[id] = threadIdx.y;
}

void printArr(int *a, int N)
{
  for (int i=0; i<N; i++) printf("%2d ", a[i]);
  printf("\n");
}

int main(int argc, char **argv)
{
  int N = atoi(argv[1]);
  for (int k=0; k<5; k++)
  {
    int *a_h, *b_h; //pointers to host memory
    int *a_d; //pointers to device memory
    int i;
    size_t size = N*sizeof(int);
    
    //allocate array on host
    a_h = (int *)malloc(size);
    b_h = (int *)malloc(size);

    //allocate array on device
    hipMalloc((void **) &a_d, size);
    
    //initialization of host data
    for (i=0; i<N; i++) a_h[i] = 0;

    //copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(int)*N, hipMemcpyHostToDevice);
    
    //do calculation on host
    dim3 dimGrid = dim3(atoi(argv[2]), atoi(argv[3]));
    dim3 dimBlock = dim3(atoi(argv[4]), atoi(argv[5]));

    if (k==0) kernelId<<<dimGrid,dimBlock>>>(a_d);
    if (k==1) kernelBlockIdxX<<<dimGrid,dimBlock>>>(a_d);
    if (k==2) kernelBlockIdxY<<<dimGrid,dimBlock>>>(a_d);
    if (k==3) kernelThreadIdxX<<<dimGrid,dimBlock>>>(a_d);
    if (k==4) kernelThreadIdxY<<<dimGrid,dimBlock>>>(a_d);

    //retrieve result from device and store in b_h
    hipMemcpy(b_h, a_d, sizeof(int)*N, hipMemcpyDeviceToHost);
    
    //print out the result
    printArr(b_h, N);

    //cleanup
    free(a_h); free(b_h); hipFree(a_d);
  }
}
