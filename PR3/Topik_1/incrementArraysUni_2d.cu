// incrementArray.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void incrementArrayOnHost(float *a, unsigned long N)
{
  unsigned long i;
  for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, unsigned long N)
{
  unsigned long idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned long idy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned long id  = idy*gridDim.x*blockDim.x + idx;
  if (id<N) a[id] = a[id]+1.f;
}

int main(int argc, char **argv)
{
  float *a, *b;
  unsigned long i, N = strtoul(argv[1], NULL, 10);
  size_t size = N*sizeof(float);

  gpuErrchk( hipMallocManaged((void **) &a, size) );
  gpuErrchk( hipMallocManaged((void **) &b, size) );

  for (i=0; i<N; i++)
  {
    a[i] = (float)i;
    b[i] = (float)i;
  }
  
  // do calculation on host
  incrementArrayOnHost(a, N);

  // do calculation on device:
  // Part 1 of 2. Compute execution configuration
  unsigned long blockDimX = strtoul(argv[2], NULL, 10);
  unsigned long blockDimY = strtoul(argv[3], NULL, 10);
  unsigned long gridDimX = strtoul(argv[4], NULL, 10);
  unsigned long gridDimY = strtoul(argv[5], NULL, 10);

  dim3 blockSize = dim3(blockDimX, blockDimY);
  dim3 gridSize = dim3(gridDimX, gridDimY);
  printf("blockDim: (%lu,%lu), gridDim: (%lu,%lu)\n", blockDimX, blockDimY, gridDimX, gridDimY);

  // Part 2 of 2. Call incrementArrayOnDevice kernel
  incrementArrayOnDevice <<< gridSize, blockSize >>> (b, N);
  hipDeviceSynchronize();
  
  // check results
  for (i=0; i<N; i++) assert(a[i] == b[i]);

  // cleanup
  hipFree(a); hipFree(b);
}
