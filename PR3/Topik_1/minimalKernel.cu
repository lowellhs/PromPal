#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void kernelId(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = idx;
}

__global__ void kernelBlockIdx(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = blockIdx.x;
}

__global__ void kernelThreadIdx(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = threadIdx.x;
}

int main(int argc, char **argv)
{
  int N = atoi(argv[1]);
  for (int k=0; k<3; k++)
  {
    int *a_h, *b_h; //pointers to host memory
    int *a_d; //pointers to device memory
    int i;
    size_t size = N*sizeof(int);
   
    //allocate array on host
    a_h = (int *)malloc(size);
    b_h = (int *)malloc(size);

    //allocate array on device
    hipMalloc((void **) &a_d, size);
  
    //initialization of host data
    for (i=0; i<N; i++) a_h[i] = 0;

    //copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(int)*N, hipMemcpyHostToDevice);
  
    //do calculation on host
    int nBlocks = atoi(argv[2]);
    int blockSize = atoi(argv[3]);
    if (k==0) { kernelId<<<nBlocks,blockSize>>>(a_d); printf("%s      :", "a[i]"); }
    if (k==1) { kernelBlockIdx<<<nBlocks,blockSize>>>(a_d); printf("%s  :", "blockIdx"); }
    if (k==2) { kernelThreadIdx<<<nBlocks,blockSize>>>(a_d); printf("%s :", "threadIdx"); }

    //retrieve result from device and store in b_h
    hipMemcpy(b_h, a_d, sizeof(int)*N, hipMemcpyDeviceToHost);
  
    //print out the result
    for (i=0; i<N; i++) printf("%2d ", b_h[i]);
    printf("\n");

    //cleanup
    free(a_h); free(b_h); hipFree(a_d);
  }
}
