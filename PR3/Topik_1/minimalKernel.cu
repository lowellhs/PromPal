#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void kernelId(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = idx;
}

__global__ void kernelBlockIdx(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = blockIdx.x;
}

__global__ void kernelThreadIdx(int *a)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[idx] = threadIdx.x;
}

int main(void)
{
  int *a_h, *b_h; //pointers to host memory
  int *a_d; //pointers to device memory
  int i;
  int N = 20;
  size_t size = N*sizeof(int);
   
  //allocate array on host
  a_h = (int *)malloc(size);
  b_h = (int *)malloc(size);

  //allocate array on device
  hipMalloc((void **) &a_d, size);
  
  //initialization of host data
  for (i=0; i<N; i++) a_h[i] = 0;

  //copy data from host to device
  hipMemcpy(a_d, a_h, sizeof(int)*N, hipMemcpyHostToDevice);
  
  //do calculation on host
  //kernelId<<<5,4>>>(a_d);
  //kernelBlockIdx<<<10,2>>>(a_d);
  kernelThreadIdx<<<10,2>>>(a_d);

  //retrieve result from device and store in b_h
  hipMemcpy(b_h, a_d, sizeof(int)*N, hipMemcpyDeviceToHost);
  
  //print out the result
  for (i=0; i<N; i++) printf("%d", b_h[i]);
  printf("\n");

  //cleanup
  free(a_h); free(b_h); hipFree(a_d);
}
